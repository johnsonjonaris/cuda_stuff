#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include "stdio.h"
#include "thrust/device_ptr.h"
#include "thrust/extrema.h"
#include "thrust/reduce.h"

int block = 16;
#define FLOAT_MAX 1e+37

__global__
void min_reduce(const float* const in, int nElements, float *d_min)
{
	extern __shared__ float shared[];
	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = FLOAT_MAX;

	if (gid < nElements)
		shared[tid] = in[gid];
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
		if (tid < s && gid < nElements)
			shared[tid] = min(shared[tid], shared[tid + s]);
		__syncthreads();
	}

	if (gid == 0)
		d_min[blockIdx.x] = shared[0];
}

float getMin(const float * const data, int nElements)
{
	const int numThreads = block*block;
	const int numBlocks = nElements/numThreads;
	unsigned int sharedSize = numThreads*sizeof(float);
	float *partial, *d_result;
	checkCudaErrors(hipMalloc(&partial, numBlocks*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_result, sizeof(float)));

	min_reduce<<<numBlocks,numThreads,sharedSize>>>(data, nElements, partial);
	min_reduce<<<1,numThreads,sharedSize>>>(partial, numBlocks, d_result);
	float h_result;
	checkCudaErrors(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(partial));
	checkCudaErrors(hipFree(d_result));
	return h_result;
}

__global__
void max_reduce(const float* const in, int nElements, float *d_max)
{
	extern __shared__ float shared[];
	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = -FLOAT_MAX;

	if (gid < nElements)
		shared[tid] = in[gid];
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
		if (tid < s && gid < nElements)
			shared[tid] = max(shared[tid], shared[tid + s]);
		__syncthreads();
	}

	if (gid == 0)
		d_max[blockIdx.x] = shared[0];
}

float getMax(const float * const data, int nElements)
{
	const int numThreads = block*block;
	const int numBlocks = nElements/numThreads;
	printf("nElements = %d, nThreads = %d, nBlocks = %d\n", nElements, numThreads, numBlocks);
	unsigned int sharedSize = numThreads*sizeof(float);
	float *partial, *d_result;
	checkCudaErrors(hipMalloc(&partial, numBlocks*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_result, sizeof(float)));

	max_reduce<<<numBlocks,numThreads,sharedSize>>>(data, nElements, partial);
	max_reduce<<<1,numThreads,sharedSize>>>(partial, numBlocks, d_result);
	float h_result;
	checkCudaErrors(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(partial));
	checkCudaErrors(hipFree(d_result));
	return h_result;
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
								  unsigned int* const d_cdf,
								  float &min_logLum,
								  float &max_logLum,
								  const size_t numRows,
								  const size_t numCols,
								  const size_t numBins)
{
	//TODO
	/*Here are the steps you need to implement
	1) find the minimum and maximum value in the input logLuminance channel
	   store in min_logLum and max_logLum
	2) subtract them to find the range
	3) generate a histogram of all the values in the logLuminance channel using
	   the formula: bin = (lum[i] - lumMin) / lumRange * numBins
	4) Perform an exclusive scan (prefix sum) on the histogram to get
	   the cumulative distribution of luminance values (this should go in the
	   incoming d_cdf pointer which already has been allocated for you)       */
	printf("(nRows, nCols) = (%d, %d)\n", numRows, numCols);
	int nPixels = numRows*numCols;
	min_logLum = getMin(d_logLuminance, nPixels);
	max_logLum = getMax(d_logLuminance, nPixels);
	//const thrust::device_ptr<const float> d_ptr = thrust::device_pointer_cast(d_logLuminance);
	//min_logLum = thrust::min_element(d_ptr, d_ptr+nPixels)[0];
	//max_logLum = thrust::max_element(d_ptr, d_ptr+nPixels)[0];
	printf("GPU: min %f, max %f\n", min_logLum, max_logLum);
	float lumRange = max_logLum - min_logLum;


}
